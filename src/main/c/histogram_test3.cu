#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>

#define numBins 10
#define width 1000
#define height 1000

#define dimBlockx 16
#define dimBlocky 16
#define blockSize dimBlockx*dimBlocky

#define dimThreadx 16
#define dimThready 16

// unique index for each thread within its block
#define tid (threadIdx.x + threadIdx.y * blockDim.y)
// unique index for each block
#define bid (blockIdx.x + blockIdx.y * gridDim.y)
// i and j are indices into the whole texture for this thread
#define tidx (( threadIdx.x + blockIdx.x * blockDim.x ) * dimThreadx)
#define tidy (( threadIdx.y + blockIdx.y * blockDim.y ) * dimThready)

hipArray *cuArray;
float* imageData;
int* dBins;
int* hBins;

int gridX;
int gridY;
int sizeBins;

// constant memory
__constant__ int* bins;
__constant__ float minZ;
__constant__ float maxZ;
__constant__ float stepZ;

// a reference to a 2D texture where each texture element contains a 1D float value
// hipReadModeElementType specifies that the returned data value should not be normalized
texture<float,  2, hipReadModeElementType> texture_float_2D;

// clamp
inline __device__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}

// bins     global memory vector to be filled with bin counts
// nbins    size of bins vector
// minX     the minimum x texture coordinate
// stepX    step size in x in texture coordinates
// minY     the minimum y texture coordinate
// stepY    step size in y in texture coordinates
// minZ     data value of the left edge of the left-most bin
// maxZ     data value of the right edge of the right-most bin
extern "C" __global__ void calculateHistogram2( float minX, float stepX,
                                                float minY, float stepY )
{
    // allocate enough shared memory for each thread to
    // have its own set of histogram bins
    __shared__ int localBins[numBins*dimBlockx*dimBlocky];

    // clear the shared memory bins (only the first numBins threads)
    int k;
    for ( k = 0 ; k < numBins ; k++ ) 
    {
        localBins[dimBlockx*dimBlocky*k + tid] = 0;
    }

    for ( int dx = 0 ; dx < dimThreadx ; dx++ )
    {
        for ( int dy = 0 ; dy < dimThready ; dy++ )
        {
            // don't over count if texture coordinates are out of bounds
            if ( dx + tidx < width && dy + tidy < height )
            {
                // perform texture lookup
                // convert block/thread ids into texture coordinates
                float x = minX + stepX * (tidx+dx);
                float y = minY + stepY * (tidy+dy);
                float result = tex2D(texture_float_2D, x, y);
    
                // calculate bin index
                float fbinIndex = floor( ( result - minZ ) / stepZ );
                int binIndex = (int) clamp( floor( ( result - minZ ) / stepZ ), 0, numBins-1 );
    
                // no need for atomic operations because each thread
                // is now building its own sub-histogram
                localBins[blockSize*binIndex + tid] += 1;
            }
        }
    }

    // wait for all threads in this block to finish
    // building their sub-histogram
    __syncthreads();

    // perform a tree reduction to combine the
    // sub-histograms on each thread into a single block-histogram
    for ( int offset = blockSize >> 1 ; offset > 0 ; offset = offset >> 1 )
    {
        if ( tid < offset )
        {
            for ( k = 0 ; k < numBins ; k++ )
            {
                localBins[blockSize*k+tid] += localBins[blockSize*k+tid+offset];
            }
        }

        // synchronize after each tree reduction step
        __syncthreads();
    }

    // at this point, the bin counts for the entire block are in
    // the first numBins entries of localBins
    // now write those bins to global memory
    if ( tid < numBins )
    {
        bins[bid*numBins+tid] = localBins[tid*blockSize];
    }
}

void initImageData( float* data )
{
    int w,h;

    float pi = atan(1) * 4;

    for ( w = 0; w < width; w++ )
    {
        for ( h = 0; h < height; h++ )
        {
            float x = w / ( float ) width;
            float y = h / ( float ) height;

            float r = rand() / (float) RAND_MAX;
            data[h+w*height] = ( y * y + sin( 2 * pi * x * x ) + r ) * 100;
        }
    }
}

void init(int argc, char **argv)
{
    // size of texture data
    unsigned int size = width * height * sizeof(float);

    // allocate space for texture data and initialize with interesting function
    imageData = (float*) malloc( size );
    initImageData( imageData );

    // set up CUDA texture description (32 bit float)
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    // create a CUDA array for accessing texture data
    hipMallocArray(&cuArray,&channelDesc,width,height);

    // copy image data from the host into the CUDA array
    hipMemcpyToArray(cuArray, 0, 0, imageData, size, hipMemcpyHostToDevice);

    // set texture access modes for the CUDA texture variable
    // (clamp access for texture coordinates outside 0 to 1)
    texture_float_2D.addressMode[0] = hipAddressModeClamp;
    texture_float_2D.addressMode[1] = hipAddressModeClamp;
    texture_float_2D.filterMode = hipFilterModeLinear;
    texture_float_2D.normalized = true;    // access with normalized texture coordinates

    // bind the array to the texture
    hipBindTextureToArray(texture_float_2D, cuArray, channelDesc);
    
    // calculate block and grid dimensions
    gridX = ceil( width / (float) (dimBlockx*dimThreadx) );
    gridY = ceil( height / (float) (dimBlocky*dimThready) );

    // allocate space for histogram bin results
    // we allocate a set of bins *for each block*
    sizeBins = sizeof( int ) * numBins * gridX * gridY;
    hBins = (int*) malloc( sizeBins );
    hipMalloc( &dBins, sizeBins );

    // copy constants to symbol/constant gpu memory
    float hminZ = -50.0;
    float hmaxZ = 200.0;
    float hstepZ = ( hmaxZ - hminZ ) / numBins;
    hipMemcpyToSymbol(HIP_SYMBOL( minZ), &hminZ, sizeof( float ) );
    hipMemcpyToSymbol(HIP_SYMBOL( maxZ), &hmaxZ, sizeof( float ) );
    hipMemcpyToSymbol(HIP_SYMBOL( stepZ), &hstepZ, sizeof( float ) );
    hipMemcpyToSymbol(HIP_SYMBOL( bins), &dBins, sizeof( int* ) ); 
}

void calculateHistogram(void)
{
    // clearing global memory is no longer necessary
    // because the values placed here are copied from
    // shared memory in the kernel (however, those shared
    // memory locations must be zeroed out)
    //hipMemset( dBins, 0, sizeBins );

    // calculate block and grid dimensions
    dim3 dimBlock( dimBlockx, dimBlocky, 1);
    dim3 dimGrid( gridX, gridY, 1);

    // run the kernel over the whole texture
    float stepX = 1.0 / width;
    float stepY = 1.0 / height;
    calculateHistogram2<<<dimGrid, dimBlock>>>( 0, stepX, 0, stepY );

    // copy results back to host
    hipMemcpy( hBins, dBins, sizeBins, hipMemcpyDeviceToHost );

    // allocate space for accumulated bin counts
    int finalBins[numBins];
    int i,j,k;
    for ( i = 0 ; i < numBins ; i++ )
    {
        finalBins[i] = 0;
    }

    clock_t start = clock();

    // collate results from each gpu block on cpu
    for ( i = 0 ; i < gridX ; i++ )
    {
        for ( j = 0 ; j < gridY ; j++ )
        {
            int block_id = ( i + j * gridY ) * numBins;
            for ( k = 0 ; k < numBins ; k++ )
            {
                finalBins[k] += hBins[block_id+k];
            }
        }
    }

    // time the cpu step
    clock_t diff = clock() - start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf( "%d millis\n", msec );

    // print results
    int sum = 0;
    for ( i = 0 ; i < numBins ; i++ )
    {
        sum += finalBins[i];
        printf( "%d\n", finalBins[i] );
    }

    printf( "sum %d\n", sum );
}

//Main program
int main(int argc, char **argv)
{
  printf("CUDA Histogram Calculator\n");

  init( argc, argv );

  calculateHistogram( );

  free( hBins );
  free( imageData );

  hipFree(dBins);
  hipFreeArray(cuArray);

  return 0;
}
