#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>

#define numBins 10
#define width 1000
#define height 1000

#define dimBlockx 16
#define dimBlocky 16

hipArray *cuArray;
float* imageData;
int* dBins;
int* hBins;

int gridX;
int gridY;
int sizeBins;

// a reference to a 2D texture where each texture element contains a 1D float value
// hipReadModeElementType specifies that the returned data value should not be normalized
texture<float,  2, hipReadModeElementType> texture_float_2D;

// clamp
inline __device__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}

// bins     global memory vector to be filled with bin counts
// nbins    size of bins vector
// minX     the minimum x texture coordinate
// stepX    step size in x in texture coordinates
// minY     the minimum y texture coordinate
// stepY    step size in y in texture coordinates
// minZ     data value of the left edge of the left-most bin
// maxZ     data value of the right edge of the right-most bin
extern "C" __global__ void calculateHistogram2( int* bins,
                                                float minX, float stepX,
                                                float minY, float stepY,
                                                float minZ, float maxZ )
{
    __shared__ int localBins[numBins];

    int tid = threadIdx.x + threadIdx.y * blockDim.y;
    int bid = blockIdx.x + blockIdx.y * gridDim.y;

    // use block and thread ids to get texture coordinates for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // convert block/thread ids into texture coordinates
    float x = minX + stepX * i;
    float y = minY + stepY * j;

    // clear the shared memory bins (only the first numBins threads)
    if ( tid < numBins )
    {
        localBins[tid] = 0;
    }

    // don't over count if texture coordinates are out of bounds
    if ( x < 1.0 && y < 1.0 )
    {
        // perform texture lookup
        float result = tex2D(texture_float_2D, x, y);
    
        // calculate bin index
        float stepZ = ( maxZ - minZ ) / numBins;
        float fbinIndex = floor( ( result - minZ ) / stepZ );
        int binIndex = (int) clamp( fbinIndex, 0, numBins-1 );
    
        // atomically add one to the bin corresponding to the data value
        atomicAdd( localBins+binIndex, 1 );
    }

    // wait for all threads in this block to finish incrementing their bin
    __syncthreads();

    // the first numBins threads each write out one shared bin to global memory
    if ( tid < numBins )
    {
        bins[bid*numBins+tid] = localBins[tid];
    }
}

void initImageData( float* data )
{
    int w,h;

    float pi = atan(1) * 4;

    for ( w = 0; w < width; w++ )
    {
        for ( h = 0; h < height; h++ )
        {
            float x = w / ( float ) width;
            float y = h / ( float ) height;

            float r = rand() / (float) RAND_MAX;
            data[h+w*height] = ( y * y + sin( 2 * pi * x * x ) + r ) * 100;
        }
    }
}

void init(int argc, char **argv)
{
    // size of texture data
    unsigned int size = width * height * sizeof(float);

    // allocate space for texture data and initialize with interesting function
    imageData = (float*) malloc( size );
    initImageData( imageData );

    // set up CUDA texture description (32 bit float)
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    // create a CUDA array for accessing texture data
    hipMallocArray(&cuArray,&channelDesc,width,height);

    // copy image data from the host into the CUDA array
    hipMemcpyToArray(cuArray, 0, 0, imageData, size, hipMemcpyHostToDevice);

    // set texture access modes for the CUDA texture variable
    // (clamp access for texture coordinates outside 0 to 1)
    texture_float_2D.addressMode[0] = hipAddressModeClamp;
    texture_float_2D.addressMode[1] = hipAddressModeClamp;
    texture_float_2D.filterMode = hipFilterModeLinear;
    texture_float_2D.normalized = true;    // access with normalized texture coordinates

    // bind the array to the texture
    hipBindTextureToArray(texture_float_2D, cuArray, channelDesc);
    
    // calculate block and grid dimensions
    gridX = ceil( width / (float) dimBlockx );
    gridY = ceil( height / (float) dimBlocky );

    // allocate space for histogram bin results
    // we allocate a set of bins *for each block*
    sizeBins = sizeof( int ) * numBins * gridX * gridY;
    hBins = (int*) malloc( sizeBins );
    hipMalloc( &dBins, sizeBins );
}

void calculateHistogram(void)
{
    // clearing global memory is no longer necessary
    // because the values placed here are copied from
    // shared memory in the kernel (however, those shared
    // memory locations must be zeroed out)
    //hipMemset( dBins, 0, sizeBins );

    // calculate block and grid dimensions
    dim3 dimBlock( dimBlockx, dimBlocky, 1);
    dim3 dimGrid( gridX, gridY, 1);

    // run the kernel over the whole texture
    float stepX = 1.0 / width;
    float stepY = 1.0 / height;
    float minZ = -50.0;
    float maxZ = 200.0;
    calculateHistogram2<<<dimGrid, dimBlock>>>( dBins, 0, stepX, 0, stepY, minZ, maxZ );

    // copy results back to host
    hipMemcpy( hBins, dBins, sizeBins, hipMemcpyDeviceToHost );

    // allocate space for accumulated bin counts
    int finalBins[numBins];
    int i,j,k;
    for ( i = 0 ; i < numBins ; i++ )
    {
        finalBins[i] = 0;
    }

    // collate results from each gpu block on cpu
    for ( i = 0 ; i < gridX ; i++ )
    {
        for ( j = 0 ; j < gridY ; j++ )
        {
            int bid = i + j * gridY;
            for ( k = 0 ; k < numBins ; k++ )
            {
                printf( "%d %d %d %d\n", i, j, k, hBins[bid+k] );
                finalBins[k] += hBins[bid+k];
            }
        }
    }

    // print results
    for ( i = 0 ; i < numBins ; i++ )
    {
        printf( "%d\n", finalBins[i] );
    }
}

//Main program
int main(int argc, char **argv)
{
  printf("CUDA Histogram Calculator\n");

  init( argc, argv );

  calculateHistogram( );

  free( hBins );
  free( imageData );

  hipFree(dBins);
  hipFreeArray(cuArray);

  return 0;
}
